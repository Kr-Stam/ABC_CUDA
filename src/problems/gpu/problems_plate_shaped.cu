#include "hip/hip_runtime.h"
#include "problems_plate_shaped.cuh"
#include <math.h>

__device__ double problems::gpu::booth(double* args, int n)
{
	if(n < 2) return 0;

	double tmp1 = (args[0] + 2*args[1] - 7);
	double tmp2 = (2*args[0] + args[1] - 5);
	return tmp1*tmp1 + tmp2*tmp2;
}

__device__ double problems::gpu::matyas(double* args, int n)
{
	if (n < 2) return 0;

	return 0.26*(args[0]*args[0] + args[1]*args[1]) -
	       0.48*args[0]*args[1];
}

__device__ double problems::gpu::mccormick(double* args, int n)
{
	if (n < 2) return 0;

	double tmp = args[0]-args[1];
	return std::sin(args[0]+args[1])+tmp*tmp -
	       1.5*args[0] + 2.5*args[1] + 1;
}

__device__ double problems::gpu::power_sum(double* args, int n, double* b)
{
	double result = 0;
	for(int i = 0; i < n; i++)
	{
		double tmp = -b[i];
		for(int j = 0; j < n; j++)
		{
			tmp += std::pow(args[j], i+1);
		}
		result += tmp*tmp;
	}
	return result;
}

__device__ double problems::gpu::power_sum2(double* args, int n)
{
	double b[] = {8, 18, 44, 114};

	return problems::gpu::power_sum(args, n, b);
}

__device__ double problems::gpu::zakharov(double* args, int n)
{
	double sum1 = 0;
	double sum2 = 0;

	for(int i = 0; i < n; i++)
	{
		sum1 += args[i]*args[i];
		sum2 += 0.5*(i+1)*args[i];
	}

	sum2 = sum2 * sum2;
	return sum1 + sum2 + sum2*sum2;
}
