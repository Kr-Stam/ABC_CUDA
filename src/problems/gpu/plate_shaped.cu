#include "hip/hip_runtime.h"
#include "plate_shaped.cuh"
#include <math.h>

__host__ __device__ float problems::gpu::booth(float* args, int n)
{
	if(n < 2) return 0;

	float tmp1 = (args[0] + 2*args[1] - 7);
	float tmp2 = (2*args[0] + args[1] - 5);
	return tmp1*tmp1 + tmp2*tmp2;
}

__host__ __device__ float problems::gpu::matyas(float* args, int n)
{
	if (n < 2) return 0;

	return 0.26*(args[0]*args[0] + args[1]*args[1]) -
	       0.48*args[0]*args[1];
}

__host__ __device__ float problems::gpu::mccormick(float* args, int n)
{
	if (n < 2) return 0;

	float tmp = args[0]-args[1];
	return std::sin(args[0]+args[1])+tmp*tmp -
	       1.5*args[0] + 2.5*args[1] + 1;
}

__host__ __device__ float problems::gpu::power_sum(float* args, int n, float* b)
{
	float result = 0;
	for(int i = 0; i < n; i++)
	{
		float tmp = -b[i];
		for(int j = 0; j < n; j++)
		{
			tmp += std::pow(args[j], i+1);
		}
		result += tmp*tmp;
	}
	return result;
}

__host__ __device__ float problems::gpu::power_sum2(float* args, int n)
{
	float b[] = {8, 18, 44, 114};

	return problems::gpu::power_sum(args, n, b);
}

__host__ __device__ float problems::gpu::zakharov(float* args, int n)
{
	float sum1 = 0;
	float sum2 = 0;

	for(int i = 0; i < n; i++)
	{
		sum1 += args[i]*args[i];
		sum2 += 0.5*(i+1)*args[i];
	}

	sum2 = sum2 * sum2;
	return sum1 + sum2 + sum2*sum2;
}
